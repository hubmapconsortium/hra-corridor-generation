#include "hip/hip_runtime.h"
#include "corridor_with_spatial_index.cuh"


void test_corridor_with_spatial_index_gpu(AATissue &example_tissue, std::vector<std::pair<int, float>> &result, std::vector<myspatial::AABBTree*> &p_aabbtrees)
{

	int n_collisions = result.size();
	myspatial::AABBTreeCUDA *h_aabbtrees = (myspatial::AABBTreeCUDA*) malloc(n_collisions * sizeof(myspatial::AABBTreeCUDA));

	for (int i = 0; i < n_collisions; i++)
	{
		auto j = result[i].first;
		myspatial::AABBTree *p_aabbtree = p_aabbtrees[j];
		
		std::vector<myspatial::Node> &node_pool = p_aabbtree->node_pool;
		std::vector<myspatial::Triangle> &triangles = p_aabbtree->triangles_;

		myspatial::Node *h_node_pool = node_pool.data();
		myspatial::Triangle *h_triangles = triangles.data();

		int n_nodes = node_pool.size();
		int n_triangles = triangles.size();

		hipMalloc(&(h_aabbtrees[i].nodes_), n_nodes * sizeof(myspatial::Node));
		hipMalloc(&(h_aabbtrees[i].triangles_), n_triangles * sizeof(myspatial::Triangle));

		hipMemcpy(h_aabbtrees[i].nodes_, h_node_pool,  n_nodes * sizeof(myspatial::Node), hipMemcpyHostToDevice);
		hipMemcpy(h_aabbtrees[i].triangles_, h_triangles,  n_triangles * sizeof(myspatial::Triangle), hipMemcpyHostToDevice);

		h_aabbtrees[i].n_nodes = n_nodes;
		h_aabbtrees[i].n_triangles = n_triangles;

	}

	myspatial::AABBTreeCUDA* d_aabbtrees;
	hipMalloc((void**)&d_aabbtrees, n_collisions * sizeof(myspatial::AABBTreeCUDA));
	hipMemcpy(d_aabbtrees, h_aabbtrees, n_collisions * sizeof(myspatial::AABBTreeCUDA), hipMemcpyHostToDevice);



	//Define Grid Configuration
	// dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	// dim3 dimGrid(MAX_BLOCK_NUMBERS, MAX_BLOCK_NUMBERS, MAX_BLOCK_NUMBERS);
	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(10, 1, 1);

    //Launch corridor GPU
    GpuTimer timer;
	timer.Start();

	test_gpu_spatial_index<<<dimGrid, dimBlock>>>(d_aabbtrees, n_collisions);

	hipDeviceSynchronize();
    print_if_cuda_error(__LINE__);
	timer.Stop();
	printf("\t\nKernel Time: %f msecs.\n", timer.Elapsed());


}

__global__ void test_gpu_spatial_index(myspatial::AABBTreeCUDA* d_aabbtrees, int n_aabbtrees)
{
	int x = blockIdx.x;
    int y = blockIdx.y;
    int z = blockIdx.z;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int k = threadIdx.z;

	// printf("x: %d, y: %d, z:%d\n", x, y, z);

	if (x < n_aabbtrees)
	{
		myspatial::AABBTreeCUDA &aabbtree = d_aabbtrees[x];
		myspatial::Triangle t = aabbtree.triangles_[0];
		int n_nodes = aabbtree.n_nodes;
		int n_triangles = aabbtree.n_triangles;

		// printf("x: %d, nodes: %p, triangles: %p\n", x, (void*) aabbtree.nodes_, (void*) aabbtree.triangles_);
		printf("nodes: %d, triangles: %d, %f, %f, %f\n", n_nodes, n_triangles, t.p1.x, t.p1.y, t.p1.z);
	}

}

// __global__ void compute_corridor_GPU_with_spatial_index(myspatial::AABBTreeCUDA* d_aabbtrees, float* target_intersection_pctgs, uint n_meshes, 
//                                         float intersect_x_min, float intersect_y_min, float intersect_z_min,
//                                         float intersect_x_max, float intersect_y_max, float intersect_z_max,
//                                         float example_d_x, float example_d_y, float example_d_z,
//                                         float step_x, float step_y, float step_z, int resolution, float tolerance,
// 										ResultContainer *result_container)
// {
    
//     // // n_meshes is the number of collided meshes

//     int x = blockIdx.x;
//     int y = blockIdx.y;
//     int z = blockIdx.z;
// 	int i = threadIdx.x;
// 	int j = threadIdx.y;
// 	int k = threadIdx.z;
	
// 	// printf("block index: %d %d %d %d %d %d\n", x, y, z, i, j, k);
	
// 	if (x >= 40 || y >= 40 || z >= 40) return;

// 	// __shared__ float intersection_volumes[10];
// 	__shared__ float intersection_volumes[10][1000];
// 	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
// 	{
// 		// for (int i = 0; i < 10; i++) intersection_volumes[i] = 0.0;
// 		for (int i = 0; i < 10; i++)
// 			for (int j = 0; j < 1000; j++)
// 			intersection_volumes[i][j] = 0.0;
// 	}
// 	__syncthreads();

// 	float delta_x = example_d_x / resolution, delta_y = example_d_y / resolution, delta_z = example_d_z / resolution;
	
// 	// center of the tissue
// 	float c_x = intersect_x_min - example_d_x / 2 + step_x * x;
// 	float c_y = intersect_y_min - example_d_y / 2 + step_y * y;
// 	float c_z = intersect_z_min - example_d_z / 2 + step_z * z;

// 	// min of the tissue
// 	float min_x = c_x - example_d_x / 2;
// 	float min_y = c_y - example_d_y / 2;
// 	float min_z = c_z - example_d_z / 2;
	
// 	int v = 0;
// 	// printf("%d", count);
	
// 	float point_c_x = min_x + (i + 0.5) * delta_x;
// 	float point_c_y = min_y + (j + 0.5) * delta_y;
// 	float point_c_z = min_z + (k + 0.5) * delta_z;
// 	float3 p = make_float3(point_c_x, point_c_y, point_c_z);
				
// 	for (int m_idx = 0; m_idx < n_meshes; m_idx++)
// 	{
// 		v = point_in_polyhedron(p, meshes, offset, m_idx);
// 		// atomicAdd(&intersection_volumes[m_idx], v);
// 		intersection_volumes[m_idx][i*10*10 + j*10 + k] = v;
// 	}

// 	__syncthreads();

// 	if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
// 	{
// 		bool is_in_corridor = true;
// 		int total_voxels = resolution * resolution * resolution;
// 		for (int m_idx = 0; m_idx < n_meshes; m_idx++)
// 		{
// 			float temp_intersect_volume = 0;
// 			for (int p_idx = 0; p_idx < 1000; p_idx++)
// 			{
// 				temp_intersect_volume += intersection_volumes[m_idx][p_idx];
// 			}

// 			// float cur_pctg = intersection_volumes[m_idx] / total_voxels;
// 			float cur_pctg = 1.0 * temp_intersect_volume / total_voxels;
// 			// printf("cur_pctg: %f\n", cur_pctg);
// 			if (cur_pctg > target_intersection_pctgs[m_idx] * (1 + tolerance) || cur_pctg < target_intersection_pctgs[m_idx] * (1 - tolerance))
// 			{
// 				is_in_corridor = false;
// 				break;
// 			}
// 		}

// 		int idx = x*40*40 + y*40 + z;
// 		if (is_in_corridor)
// 		{
// 			float3 tissue_center = make_float3(c_x, c_y, c_z);
// 			result_container->corridor_array[idx] = tissue_center;
// 			result_container->point_is_in_corridor_array[idx] = true;
// 		}
// 		else{
// 			result_container->point_is_in_corridor_array[idx] = false;
// 		}

// 	}
// }